
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define GRADIENT_SIZE 16

#define C_X_MIN -0.188
#define C_X_MAX -0.012
#define C_Y_MIN 0.554
#define C_Y_MAX 0.754

#define IMAGE_SIZE 4096
#define ARRAY_SIZE (3 * IMAGE_SIZE * IMAGE_SIZE * sizeof(unsigned char)) 

#define PIXEL_WIDTH ((C_X_MAX - C_X_MIN) / IMAGE_SIZE)
#define PIXEL_HEIGHT ((C_Y_MAX - C_Y_MIN) / IMAGE_SIZE)
#define ITERATION_MAX 200

int colors[51] = {
                    66, 30, 15,
                    25, 7, 26,
                    9, 1, 47,
                    4, 4, 73,
                    0, 7, 100,
                    12, 44, 138,
                    24, 82, 177,
                    57, 125, 209,
                    134, 181, 229,
                    211, 236, 248,
                    241, 233, 191,
                    248, 201, 95,
                    255, 170, 0,
                    204, 128, 0,
                    153, 87, 0,
                    106, 52, 3,
                    16, 16, 16,
                    };
int *d_colors;

int x_grid;
int y_grid;
int x_block;
int y_block;
dim3 dimGrid;
dim3 dimBlock;

unsigned char *image_buffer;
unsigned char *d_image_buffer;

void init(int argc, char *argv[])
{
    if (argc != 5) {
        printf("usage: ./mandelbrot_cu x_grid y_grid x_blocks y_blocks");
        exit(0);
    } 
    sscanf(argv[1], "%lf", &x_grid);
    sscanf(argv[2], "%lf", &y_grid);
    sscanf(argv[3], "%lf", &x_block);
    sscanf(argv[4], "%lf", &y_block);

    hipHostMalloc((void **) &image_buffer, ARRAY_SIZE, hipHostMallocDefault);
    hipMalloc((void **) &d_image_buffer, ARRAY_SIZE);

    hipMalloc((void **) &d_colors, 51 * sizeof(int));
    hipMemcpy(d_colors, colors, 51 * sizeof(int), hipMemcpyHostToDevice);

    dimGrid = dim3(x_grid, y_grid, 1);
    dimBlock = dim3(x_block, y_block, 1);
}

void write_to_file()
{
    FILE * file;
    const char *filename = "output.ppm";
    const char *comment  = "# ";

    int max_color_component_value = 255;

    file = fopen(filename,"wb");

    fprintf(file, "P6\n %s\n %d\n %d\n %d\n", comment,
            IMAGE_SIZE, IMAGE_SIZE, max_color_component_value);

    for(int i = 0; i < IMAGE_SIZE * IMAGE_SIZE; i += 3){
        fwrite(&image_buffer[i], 1 , 3, file);
    };

    fclose(file);
}

__global__ void gpu_compute_mandelbrot(unsigned char *buffer, int *colors_d)
{
    double z_x = 0.0;
    double z_y = 0.0;
    double z_x_squared = 0.0;
    double z_y_squared = 0.0;
    double escape_radius_squared = 4;

    double c_x;
    double c_y;
    
    int i_y = blockIdx.y * blockDim.y + threadIdx.y; 
    int i_x = blockIdx.x * blockDim.x + threadIdx.x;

    int color;
    int iteration;

    c_y = C_Y_MIN + i_y * PIXEL_HEIGHT;
    if (fabs(c_y) < PIXEL_HEIGHT / 2)
        c_y = 0.0;
    c_x = C_X_MIN + i_x * PIXEL_WIDTH;
    for (iteration = 0;
                iteration < ITERATION_MAX && \
                ((z_x_squared + z_y_squared) < escape_radius_squared);
                iteration++) {
                z_y         = 2 * z_x * z_y + c_y;
                z_x         = z_x_squared - z_y_squared + c_x;

                z_x_squared = z_x * z_x;
                z_y_squared = z_y * z_y;
    }
    color = (iteration == ITERATION_MAX) ? GRADIENT_SIZE : iteration % GRADIENT_SIZE;
    for (int i = 0; i < 3; i++)
        buffer[(IMAGE_SIZE * i_y) + i_x + i] = colors_d[color * 3 + i];
}

void compute_mandelbrot()
{
    gpu_compute_mandelbrot<<<dimGrid, dimBlock>>>(d_image_buffer, d_colors);
    hipMemcpy(image_buffer, d_image_buffer, ARRAY_SIZE, hipMemcpyDeviceToHost);
}

int main(int argc, char *argv[])
{
    init(argc, argv);

    compute_mandelbrot();

    write_to_file();

    hipHostFree(image_buffer); hipFree(d_image_buffer);
    hipFree(d_colors);

    printf("Foi");
    return 0;
}